#include "hip/hip_runtime.h"
// #include "pdc.h"
// #include "pdc_private.h"
extern "C"
{
#include "pdc_public.h"
#include "plugin_helper.h"
}


#undef NDEBUG // enable asserts on release build
#include <assert.h>

#include <stdio.h>

#include "api.hh"

#include "cli/quality_viewer.hh"
#include "cli/timerecord_viewer.hh"
#include "utils/autotune.cuh"

#include <hip/hip_runtime.h>

#define CUDA_RUNTIME_API_CALL(apiFuncCall)                               \
{                                                                        \
  hipError_t _status = apiFuncCall;                                     \
  if (_status != hipSuccess) {                                          \
    fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n", \
      __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));    \
    exit(-1);                                                            \
  }                                                                      \
}

bool is_device_pointer(const void *ptr)
{
  struct hipPointerAttribute_t attributes;
  hipPointerGetAttributes(&attributes, ptr);
  return (attributes.devicePointer != NULL);
}

void compress_array(void *buf, int ndim, uint64_t *dims, size_t size)
{
  using T = float;
  Capsule<T> input("uncompressed");

  {
    int len = 1;
    for (int i = 0; i < ndim; i++) {
      len *= dims[i];
    }

    /* cuSZ requires a 3% overhead on device (not required on host). */
    size_t uncompressed_alloclen = len * 1.03;
    size_t decompressed_alloclen = uncompressed_alloclen;

    /* code snippet for looking at the device array easily */
    auto peek_devdata = [](T* d_arr, size_t num = 20) {
      thrust::for_each(thrust::device, d_arr, d_arr + num, [=] __device__ __host__(const T i) { printf("%f\t", i); });
      printf("\n");
    };

    T *d_uncompressed;
    CUDA_RUNTIME_API_CALL(hipMalloc(&d_uncompressed, sizeof(T) * uncompressed_alloclen ));
    CUDA_RUNTIME_API_CALL(hipMemcpy(d_uncompressed, buf, size, hipMemcpyDeviceToDevice));

    /* a casual peek */
    printf("peeking uncompressed data, 20 elements\n");
    peek_devdata(d_uncompressed, 20);

    hipStream_t stream;
    CUDA_RUNTIME_API_CALL(hipStreamCreate(&stream));

    using Compressor = typename cusz::Framework<T>::LorenzoFeaturedCompressor;
    Compressor*  compressor = new Compressor;
    BYTE* exposed_compressed;
    size_t compressed_len;
    {
      cusz::TimeRecord timerecord;
      cusz::Context* ctx = new cusz::Context();
      ctx->set_len(dims[0], 1, 1, 1)
        .set_eb(47803.025242)                      // numeric
        .set_control_string("mode=r2r");  // string

      cusz::Header header;
      cusz::core_compress(
          compressor, ctx,                             // compressor & config
          d_uncompressed, uncompressed_alloclen,       // input
          exposed_compressed, compressed_len, header,  // output
          stream, &timerecord);

      /* User can interpret the collected time information in other ways. */
      cusz::TimeRecordViewer::view_compression(&timerecord, len * sizeof(T), compressed_len);

      /* verify header */
      printf("header.%-*s : %x\n",            12, "(addr)", &header);
      printf("header.%-*s : %lu, %lu, %lu\n", 12, "{x,y,z}", header.x, header.y, header.z);
      printf("header.%-*s : %lu\n",           12, "filesize", header.get_filesize());
    }

    printf("compression done\n");
    printf("compressed_len: %ld\n", compressed_len);

    /* If needed, User should perform a memcopy to transfer `exposed_compressed` before `compressor` is destroyed. */
    BYTE* compressed;
    CUDA_RUNTIME_API_CALL(hipMalloc(&compressed, compressed_len));
    CUDA_RUNTIME_API_CALL(hipMemcpy(compressed, exposed_compressed, compressed_len, hipMemcpyDeviceToDevice));
  }
}

size_t pdc_cusz_compress_cpp(void *dataIn, pdc_var_type_t srcType, int ndim, uint64_t *dims, void **dataOut, pdc_var_type_t destType)
{
  FUNC_ENTER(NULL);
  fprintf(stdout, "\n[TRANSFORM] pdc_sz_compress_cpp\n");

  assert(srcType >= 0);
  // TODO: jjravi, if mptr is in host memory

  // CUDA_RUNTIME_API_CALL( hipSetDevice(0) );

  hipStream_t stream;
  CUDA_RUNTIME_API_CALL(hipStreamCreate(&stream));

  // hipCtx_t pctx;
  // hipCtxGetCurrent(&pctx);
  // printf("pctx: %p\n", pctx);

  int len = 1;
  for (int i = 0; i < ndim; i++) {
    len *= dims[i];
  }

  // compress_array(dataIn, ndim, dims, len*sizeof(float));

  // code snippet for looking at the device array easily
  using T = float;
  auto peek_devdata = [](T* d_arr, size_t num = 20) {
    thrust::for_each(thrust::device, d_arr, d_arr + num, [=] __device__ __host__(const T i) { printf("%f\t", i); });
    printf("\n");
  };

  // TODO: jjravi UVM is slower than device memory
  // float *d_uncompressed;
  // d_uncompressed = (float *)dataIn;
  printf("is_device_pointer: %d\n", is_device_pointer(dataIn));

  // TODO: and compressor needs 3% oveerhead?
  T *d_uncompressed;
  // cuSZ requires a 3% overhead on device (not required on host).
  size_t uncompressed_alloclen = len * 1.03;
  size_t decompressed_alloclen = uncompressed_alloclen;
  CUDA_RUNTIME_API_CALL(hipMalloc(&d_uncompressed, sizeof(T) * uncompressed_alloclen));
  // CUDA_RUNTIME_API_CALL(hipMemcpy(d_uncompressed, dataIn, sizeof(T) * len, hipMemcpyDeviceToDevice));
  CUDA_RUNTIME_API_CALL(hipMemcpy(d_uncompressed, dataIn, sizeof(T) * len, hipMemcpyHostToDevice));

  // printf("peeking uncompressed data, 20 elements\n");
  // peek_devdata(d_uncompressed, 20);

  using Compressor = typename cusz::Framework<T>::LorenzoFeaturedCompressor;
  Compressor*  compressor = new Compressor;
  BYTE* exposed_compressed;

  // const float eb = 47803.025242;
  // const float eb = 0.00001;
  const double eb = 1e-2;

  cusz::Context* ctx = new cusz::Context();
  switch(ndim)
  {
    case 1:
      printf("1d data\n");
      ctx->set_len(dims[0], 1, 1, 1)
        .set_eb(eb)                      // numeric
        .set_control_string("mode=r2r");  // string

      AutoconfigHelper::autotune(ctx);

      ctx->set_eb(eb);
      ctx->set_control_string("mode=r2r");
      ctx->verbose = true;
      ctx->mode = "r2r";
      ctx->eb = eb;
      break;
    case 2:
      printf("2d data\n");
      ctx->set_len(dims[0], dims[1], 1, 1)
        .set_eb(eb)                      // numeric
        .set_control_string("mode=abs");  // string
      break;
    case 3:
      printf("3d data\n");
      ctx->set_len(dims[0], dims[1], dims[2], 1)
        .set_eb(eb)                      // numeric
        .set_control_string("mode=abs");  // string
      break;
    case 4:
      printf("4d data\n");
      ctx->set_len(dims[0], dims[1], dims[2], dims[3])
        .set_eb(eb)                      // numeric
        .set_control_string("mode=abs");  // string
      break;
    default:
      assert(0);
      break;
  }

  cusz::TimeRecord timerecord;
  size_t compressed_len;
  cusz::Header header;

  {
    nvtxRangePush("core_compress");

    cusz::core_compress(
      compressor, ctx,                           // compressor & config
      d_uncompressed, uncompressed_alloclen,     // input
      exposed_compressed, compressed_len, header,  // output
      stream, &timerecord
    );

    nvtxRangePop();
  }


  {
    nvtxRangePush("view_compression");

    // User can interpret the collected time information in other ways
    cusz::TimeRecordViewer::view_compression(&timerecord, len * sizeof(T), compressed_len);

    printf("header.%-*s : %x\n",            12, "(addr)", &header);
    printf("header.%-*s : %lu, %lu, %lu\n", 12, "{x,y,z}", header.x, header.y, header.z);
    printf("header.%-*s : %lu\n",           12, "filesize", header.get_filesize());


    nvtxRangePop();
  }



  // *dataOut = bytes;

  // CUDA_RUNTIME_API_CALL(hipMallocManaged((void **)&dataOut, compressed_len));

  // TODO: temporarily allocate the full amount
  
  CUDA_RUNTIME_API_CALL(hipMallocManaged(dataOut, compressed_len));
  CUDA_RUNTIME_API_CALL(hipMemcpy(*dataOut, exposed_compressed, compressed_len, hipMemcpyDeviceToDevice));
  size_t outSize = compressed_len;

  fprintf(stdout, "\n[TRANSFORM] successfully compressed\n");
  fprintf(stdout, "\n[TRANSFORM] %ld bytes -> %ld bytes\n", len*sizeof(T), outSize);

  // TODO: jjravi, change return type to be bytes instead of elements?
  // outSize = (size_t)( (float)outSize / (float)PDC_get_var_type_size(srcType) );

  FUNC_LEAVE(outSize);
  return outSize;
}

extern "C"
{
  size_t pdc_cusz_compress(void *dataIn, pdc_var_type_t srcType, int ndim, uint64_t *dims, void **dataOut, pdc_var_type_t destType)
  {
    return pdc_cusz_compress_cpp(dataIn, srcType, ndim, dims, dataOut, destType);
  }
}

