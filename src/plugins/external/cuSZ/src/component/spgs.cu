/**
 * @file spgs.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.3
 * @date 2021-12-01
 * (created) 202-11-12 (rev.1) 2021-12-01
 *
 * (C) 2021 by Washington State University, Argonne National Laboratory
 *
 */

#include "../utils.hh"
#include "spgs.cuh"

namespace cusz {

template <typename T>
template <cusz::LOC FROM, cusz::LOC TO>
spGS<T>& spGS<T>::consolidate(uint8_t* dst)
{
    constexpr auto direction = CopyDirection<FROM, TO>::direction;
    auto           nbyte_idx = nnz * sizeof(int);
    auto           nbyte_val = nnz * sizeof(T);
    // index first
    CHECK_CUDA(hipMemcpy(dst, /*       */ idx, nbyte_idx, direction));
    CHECK_CUDA(hipMemcpy(dst + nbyte_idx, val, nbyte_val, direction));

    return *this;
}

template <typename T>
void spGS<T>::encode(
    T*            in,
    uint32_t      in_len,
    int*          nullarray,
    int*&         out_idx,
    T*&           out_val,
    int&          out_nnz,
    unsigned int& dump_nbyte)
{
    this->idx = out_idx;
    this->val = out_val;

    {  // phase 1: count nnz
        cuda_timer_t t;
        t.timer_start();
        out_nnz = thrust::count_if(thrust::device, in, in + in_len, [] __device__(const T& x) { return x != 0; });
        t.timer_end();
        milliseconds = t.get_time_elapsed();
    }
    // TODO improve
    this->nnz = out_nnz;

    // phase 2: gather
    thrust::counting_iterator<int> zero(0);
    using Tuple = thrust::tuple<T, int>;

    auto zipped_in      = thrust::make_zip_iterator(thrust::make_tuple(in, zero));
    auto zipped_in_end  = thrust::make_zip_iterator(thrust::make_tuple(in + in_len, zero + in_len));
    auto zipped_out     = thrust::make_zip_iterator(thrust::make_tuple(out_val, out_idx));
    auto zipped_out_end = thrust::make_zip_iterator(thrust::make_tuple(out_val + out_nnz, out_idx + out_nnz));

    {
        cuda_timer_t t;
        t.timer_start();
        thrust::copy_if(thrust::device, zipped_in, zipped_in_end, zipped_out, [] __host__ __device__(const Tuple& t) {
            return thrust::get<0>(t) != 0;
        });
        t.timer_end();
        milliseconds += t.get_time_elapsed();
    }

    dump_nbyte = (sizeof(int) + sizeof(T)) * out_nnz;
}

template <typename T>
void spGS<T>::decode(int*& in_idx, T*& in_val, int nnz, T* out)
{
    cuda_timer_t t;
    t.timer_start();
    thrust::decode(thrust::device, in_val, in_val + nnz, in_idx, out);
    t.timer_end();
    milliseconds = t.get_time_elapsed();
}

template <typename T>
void spGS<T>::decode(uint8_t* _pool, int nnz, T* out, uint32_t out_len)
{
    auto nbyte_idx = nnz * sizeof(int);
    auto in_idx    = reinterpret_cast<int*>(_pool);
    auto in_val    = reinterpret_cast<T*>(_pool + (nbyte_idx));

    decode(in_idx, in_val, nnz, out);
}

}  // namespace cusz

#define SPGS_TYPE cusz::spGS<float>

template class SPGS_TYPE;

template SPGS_TYPE& SPGS_TYPE::consolidate<cusz::LOC::HOST, cusz::LOC::HOST>(uint8_t*);
template SPGS_TYPE& SPGS_TYPE::consolidate<cusz::LOC::HOST, cusz::LOC::DEVICE>(uint8_t*);
template SPGS_TYPE& SPGS_TYPE::consolidate<cusz::LOC::DEVICE, cusz::LOC::HOST>(uint8_t*);
template SPGS_TYPE& SPGS_TYPE::consolidate<cusz::LOC::DEVICE, cusz::LOC::DEVICE>(uint8_t*);
