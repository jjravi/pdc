#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <unistd.h>
#include <time.h>
#include <math.h>
#include <cinttypes>

#include <omp.h>

#undef NDEBUG
#include <assert.h>

#include <bitset>
#include <iostream>

#include <unordered_map>
#include <vector>
#include <algorithm>

#include <sys/time.h>
#include <time.h>
#include <unistd.h>

#include <array>
#include <functional>
#include <iomanip>
#include <iostream>

#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/discard_iterator.h>

extern "C"
{
#include "pdc_public.h"
#include "plugin_helper.h"
}

#include <mpi.h>

#define CUDA_RUNTIME_API_CALL(apiFuncCall)                                  \
{                                                                           \
  hipError_t _status = apiFuncCall;                                        \
  if (_status != hipSuccess) {                                             \
    fprintf(stderr, "%s:%d: error: rt function %s failed with error %s.\n", \
        __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));       \
    exit(-1);                                                               \
  }                                                                         \
}


static double gettime_ms()
{
  struct timespec t;
  clock_gettime(CLOCK_MONOTONIC_RAW, &t);
  return (t.tv_sec+t.tv_nsec*1e-9)*1000;
}

// NOTE: caller needs to free the buffer
void readFileC(const char *filepath, uint8_t **bytes, size_t &num_bytes)
{
  FILE *pFile = fopen(filepath , "rb");
  assert(pFile != NULL);

  // obtain file size:
  if (num_bytes <= 0)
  {
    fseek (pFile, 0L, SEEK_END);
    num_bytes = ftell(pFile);
    fseek(pFile, 0L, SEEK_SET);
  }
  assert(num_bytes > 0);

  // printf("size: %ld bytes\n", num_bytes);

  // allocate memory to contain the whole file:
  // *bytes = (uint8_t*) malloc (sizeof(char)*num_bytes);
  CUDA_RUNTIME_API_CALL(hipHostAlloc(bytes, sizeof(char)*num_bytes, hipHostMallocDefault));
  // *bytes = (uint8_t*) malloc (sizeof(char)*num_bytes);
  // *bytes = (uint8_t *)calloc(num_bytes, sizeof(char));
  // assert(*bytes != NULL);

  // copy the file into the *bytes:
  size_t result = fread (*bytes,1,num_bytes,pFile);
  assert(result == num_bytes);

  fclose (pFile);
}

struct freq_calc
{
  const size_t n;
  freq_calc(size_t n) : n(n) {}

  __host__ __device__ float operator()(const int &x) const

  {
    float dx = (float)x;
    float dn = (float)n;
    float prob = dx / dn;
    return (-(prob * log2(prob)));
  }
};

// calculate shannon's entropy
// - Σ i=1 to n     P(x_i) * log P(x_i)
void shannon_entropy(float *buf, size_t nitems)
{
  double tstart = gettime_ms();
  thrust::device_vector<float> d_vbuf(nitems);

  double t30 = gettime_ms();
  CUDA_RUNTIME_API_CALL(hipMemcpy(thrust::raw_pointer_cast(d_vbuf.data()), buf, nitems*sizeof(float), hipMemcpyHostToDevice));
  double t31 = gettime_ms();
  // printf("h2d time: %lf ms\n", t31 - t30);

  double t40 = gettime_ms();
  thrust::sort(thrust::device, d_vbuf.begin(), d_vbuf.end(), thrust::greater<float>());
  double t41 = gettime_ms();
  // printf("device sort time: %lf ms\n", t41 - t40);

  double t50 = gettime_ms();
  thrust::device_vector<int> d_values(nitems, 1);
  auto new_end = thrust::reduce_by_key(thrust::device,
    d_vbuf.begin(), d_vbuf.end(), d_values.begin(),
    thrust::make_discard_iterator(), d_values.begin());
  double t51 = gettime_ms();
  // printf("device reduce_by_key time: %lf ms\n", t51 - t50);

  // prints to stdout
  // thrust::copy(d_values.begin(), d_values.begin()+10, std::ostream_iterator<float>(std::cout, ", "));
  // printf("\n");

  thrust::device_vector<int>::iterator iter1 = d_values.begin();
  int values_size = thrust::distance(iter1, new_end.second);

  double t60 = gettime_ms();
  double entropy = thrust::transform_reduce(thrust::device, d_values.begin(), new_end.second, freq_calc(nitems), 0.0f, thrust::plus<float>());
  double t61 = gettime_ms();
  // printf("transform_reduce time: %lf ms\n", t61 - t60);

  double tend = gettime_ms();
  // printf("total time: %lf ms\n", tend - tstart);
  
  int rank = 0;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  // printf("entropy: %f\n", entropy);
  printf("rank=%d, entropy=%f\n", rank, entropy);
}

void shannon_entropy_d(double *buf, size_t nitems)
{
  double tstart = gettime_ms();
  thrust::device_vector<double> d_vbuf(nitems);

  double t30 = gettime_ms();
  CUDA_RUNTIME_API_CALL(hipMemcpy(thrust::raw_pointer_cast(d_vbuf.data()), buf, nitems*sizeof(double), hipMemcpyHostToDevice));
  double t31 = gettime_ms();
  // printf("h2d time: %lf ms\n", t31 - t30);

  double t40 = gettime_ms();
  thrust::sort(thrust::device, d_vbuf.begin(), d_vbuf.end(), thrust::greater<double>());
  double t41 = gettime_ms();
  // printf("device sort time: %lf ms\n", t41 - t40);

  double t50 = gettime_ms();
  thrust::device_vector<int> d_values(nitems, 1);
  auto new_end = thrust::reduce_by_key(thrust::device,
    d_vbuf.begin(), d_vbuf.end(), d_values.begin(),
    thrust::make_discard_iterator(), d_values.begin());
  double t51 = gettime_ms();
  // printf("device reduce_by_key time: %lf ms\n", t51 - t50);

  // prints to stdout
  // thrust::copy(d_values.begin(), d_values.begin()+10, std::ostream_iterator<double>(std::cout, ", "));
  // printf("\n");

  thrust::device_vector<int>::iterator iter1 = d_values.begin();
  int values_size = thrust::distance(iter1, new_end.second);

  double t60 = gettime_ms();
  double entropy = thrust::transform_reduce(thrust::device, d_values.begin(), new_end.second, freq_calc(nitems), 0.0f, thrust::plus<double>());
  double t61 = gettime_ms();
  // printf("transform_reduce time: %lf ms\n", t61 - t60);

  double tend = gettime_ms();
  // printf("total time: %lf ms\n", tend - tstart);
  
  int rank = 0;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  // printf("entropy: %f\n", entropy);
  printf("rank=%d, entropy=%f\n", rank, entropy);
}

int main(int argc, char* argv[])
{
  if(argc != 2)  fprintf(stderr, "usage: %s <file.bin>\n", argv[0]);
  const char *filename = argv[1];

  printf("reading file: %s\n", filename);

  uint8_t *bytes;
  // size_t len = 4;
  // size_t num_bytes = len*sizeof(float);

  size_t num_bytes = 0;
  double t0 = gettime_ms();
  readFileC(filename, &bytes, num_bytes);
  double t1 = gettime_ms();
  printf("read time: %lf ms\n", t1 - t0);
  size_t len = num_bytes / sizeof(float);

  float *buf = (float *)bytes;
  shannon_entropy(buf, len);

  return EXIT_SUCCESS;
}

size_t pdc_entropy_cpp(void *dataIn, pdc_var_type_t srcType, int ndim, uint64_t *dims, void **dataOut, pdc_var_type_t destType)
{
  FUNC_ENTER(NULL);
  // fprintf(stdout, "\n[TRANSFORM] pdc_entropy_cpp\n");

  // assert(srcType >= 0);
  // CUDA_RUNTIME_API_CALL( hipSetDevice(0) );

  hipStream_t stream;
  CUDA_RUNTIME_API_CALL(hipStreamCreate(&stream));

  int len = 1;
  for (int i = 0; i < ndim; i++) {
    len *= dims[i];
  }

  void *buf = (void *)dataIn;

  switch (srcType)
  {
    case PDC_FLOAT:
      shannon_entropy((float *)buf, len);
      break;
    case PDC_DOUBLE:
      shannon_entropy_d((double *)buf, len);
      break;
    default:
      break;
  }

  destType = srcType;
  *dataOut = dataIn;

  FUNC_LEAVE(len);
  return len;
}

extern "C"
{
  size_t pdc_entropy(void *dataIn, pdc_var_type_t srcType, int ndim, uint64_t *dims, void **dataOut, pdc_var_type_t destType)
  {
    return pdc_entropy_cpp(dataIn, srcType, ndim, dims, dataOut, destType);
  }
}

