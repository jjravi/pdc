/**
 * @file
 * @author John J. Ravi (jjravi)
 *
 */

#include <mpi.h>

extern "C"
{
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <getopt.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
#include <inttypes.h>

#include <nvtx3/nvToolsExt.h>

#include "pdc.h"
#include "pdc_transform.h"

#ifdef ENABLE_MPI
#include "pdc_mpi.h"
#endif
}

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#undef NDEBUG // enable asserts on release build
#include <assert.h>

#define PDC_API_CALL(apiFuncCall)                                        \
{                                                                        \
  perr_t _status = apiFuncCall;                                          \
  if (_status != SUCCEED) {                                              \
    fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n", \
      __FILE__, __LINE__, #apiFuncCall, _status);                        \
    exit(-1);                                                            \
  }                                                                      \
}

#define CUDA_RUNTIME_API_CALL(apiFuncCall)                               \
{                                                                        \
  hipError_t _status = apiFuncCall;                                     \
  if (_status != hipSuccess) {                                          \
    fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n", \
      __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));    \
    exit(-1);                                                            \
  }                                                                      \
}

#define NPARTICLES 8388608

double
uniform_random_number()
{
  return (((double)rand()) / ((double)(RAND_MAX)));
}

int main(int argc, char **argv)
{
  int     rank = 0, size = 1;
  perr_t  ret;
#ifdef ENABLE_MPI
  MPI_Comm comm;
#else
  int comm = 1;
#endif
  int       x_dim = 64;
  int       y_dim = 64;
  int       z_dim = 64;

#ifdef ENABLE_MPI
  MPI_Init(&argc, &argv);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &size);
  MPI_Comm_dup(MPI_COMM_WORLD, &comm);
#endif

  srand(0);

  CUDA_RUNTIME_API_CALL( hipSetDevice(0) );

  // hipCtx_t pctx;
  // hipCtxGetCurrent(&pctx);
  // printf("pctx: %p\n", pctx);

  // avoids cuSZ init overhead
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);

  nvtxRangePush("vpicio");

  uint64_t numparticles = NPARTICLES;
  if (argc == 2) {
    numparticles = atoll(argv[1]);
    if (rank == 0)
      printf("Writing %" PRIu64 " number of particles with %d clients.\n", numparticles, size);
  }

  uint64_t  dims[1] = {numparticles * size};

  float *x = (float *)malloc(numparticles * sizeof(float));
  float *y = (float *)malloc(numparticles * sizeof(float));
  float *z = (float *)malloc(numparticles * sizeof(float));

  float *px = (float *)malloc(numparticles * sizeof(float));
  float *py = (float *)malloc(numparticles * sizeof(float));
  float *pz = (float *)malloc(numparticles * sizeof(float));

  int *id1 = (int *)malloc(numparticles * sizeof(int));
  int *id2 = (int *)malloc(numparticles * sizeof(int));

  float *d_x;
  float *d_y;
  float *d_z;

  float *d_px;
  float *d_py;
  float *d_pz;

  float *d_id1;
  float *d_id2;

  // CUDA_RUNTIME_API_CALL( hipMalloc((void **)&d_x, numparticles*sizeof(float)) );
  // CUDA_RUNTIME_API_CALL( hipMalloc((void **)&d_y, numparticles*sizeof(float)) );
  // CUDA_RUNTIME_API_CALL( hipMalloc((void **)&d_z, numparticles*sizeof(float)) );
  // CUDA_RUNTIME_API_CALL( hipMalloc((void **)&d_px, numparticles*sizeof(float)) );
  // CUDA_RUNTIME_API_CALL( hipMalloc((void **)&d_py, numparticles*sizeof(float)) );
  // CUDA_RUNTIME_API_CALL( hipMalloc((void **)&d_pz, numparticles*sizeof(float)) );
  // CUDA_RUNTIME_API_CALL( hipMalloc((void **)&d_id1, numparticles*sizeof(int)) );
  // CUDA_RUNTIME_API_CALL( hipMalloc((void **)&d_id2, numparticles*sizeof(int)) );

  // // compute
  // for (uint64_t i = 0; i < numparticles; i++) {
  //   id1[i] = i;
  //   id2[i] = i * 2;
  //   x[i]   = uniform_random_number() * x_dim;
  //   y[i]   = uniform_random_number() * y_dim;
  //   z[i]   = ((float)id1[i] / numparticles) * z_dim;
  //   px[i]  = uniform_random_number() * x_dim;
  //   py[i]  = uniform_random_number() * y_dim;
  //   pz[i]  = ((float)id2[i] / numparticles) * z_dim;
  // }

  CUDA_RUNTIME_API_CALL( hipMallocManaged((void **)&d_x, numparticles*sizeof(float)) );
  CUDA_RUNTIME_API_CALL( hipMallocManaged((void **)&d_y, numparticles*sizeof(float)) );
  CUDA_RUNTIME_API_CALL( hipMallocManaged((void **)&d_z, numparticles*sizeof(float)) );
  CUDA_RUNTIME_API_CALL( hipMallocManaged((void **)&d_px, numparticles*sizeof(float)) );
  CUDA_RUNTIME_API_CALL( hipMallocManaged((void **)&d_py, numparticles*sizeof(float)) );
  CUDA_RUNTIME_API_CALL( hipMallocManaged((void **)&d_pz, numparticles*sizeof(float)) );
  CUDA_RUNTIME_API_CALL( hipMallocManaged((void **)&d_id1, numparticles*sizeof(int)) );
  CUDA_RUNTIME_API_CALL( hipMallocManaged((void **)&d_id2, numparticles*sizeof(int)) );

  // compute
  for (uint64_t i = 0; i < numparticles; i++) {
    d_id1[i] = i;
    d_id2[i] = i * 2;
    // d_x[i]   = uniform_random_number() * x_dim;
    // d_x[i]   = 0.1 * i * x_dim;
    d_x[i]   = i;
    d_y[i]   = uniform_random_number() * y_dim;
    d_z[i]   = ((float)id1[i] / numparticles) * z_dim;
    d_px[i]  = uniform_random_number() * x_dim;
    d_py[i]  = uniform_random_number() * y_dim;
    d_pz[i]  = ((float)id2[i] / numparticles) * z_dim;
  }

  int* data;
  size_t len = 10;
  int fullUVMSupported = 0;
  CUDA_RUNTIME_API_CALL(hipDeviceGetAttribute(&fullUVMSupported, hipDeviceAttributeConcurrentManagedAccess, 0));

  if (fullUVMSupported) {
    // int id = -1;
    // CUDA_RUNTIME_API_CALL( hipGetDevice(&id) );
    CUDA_RUNTIME_API_CALL( hipMemPrefetchAsync(d_x,   numparticles*sizeof(float), 0) );
    CUDA_RUNTIME_API_CALL( hipMemPrefetchAsync(d_y,   numparticles*sizeof(float), 0) );
    CUDA_RUNTIME_API_CALL( hipMemPrefetchAsync(d_z,   numparticles*sizeof(float), 0) );
    CUDA_RUNTIME_API_CALL( hipMemPrefetchAsync(d_px,  numparticles*sizeof(float), 0) );
    CUDA_RUNTIME_API_CALL( hipMemPrefetchAsync(d_py,  numparticles*sizeof(float), 0) );
    CUDA_RUNTIME_API_CALL( hipMemPrefetchAsync(d_pz,  numparticles*sizeof(float), 0) );
    CUDA_RUNTIME_API_CALL( hipMemPrefetchAsync(d_id1, numparticles*sizeof(int),   0) );
    CUDA_RUNTIME_API_CALL( hipMemPrefetchAsync(d_id2, numparticles*sizeof(int),   0) );
  }

  pdcid_t pdc_id = PDCinit("pdc");

  pdcid_t container_prop = PDCprop_create(PDC_CONT_CREATE, pdc_id);
  pdcid_t container_id = PDCcont_create_col("c1", container_prop);
  pdcid_t obj_prop_xx = PDCprop_create(PDC_OBJ_CREATE, pdc_id);

  ///////////////////////////////////
  // // issue 4 RPCs (these will proceed concurrently using callbacks)
  // int req_num = 4;
  // for (int i = 0; i < req_num; i++) run_my_rpc(i);
  // printf("done issuing run_my_rpc\n");
  ///////////////////////////////////

  ////////////////////////////////////////////////////
  // PDCprop_set_obj_transfer_region_type(obj_prop_xx, PDC_OBJ_STATIC);
  PDCprop_set_obj_transfer_region_type(obj_prop_xx, PDC_REGION_STATIC);
  // PDCprop_set_obj_transfer_region_type(obj_prop_xx, PDC_REGION_DYNAMIC);
  // PDCprop_set_obj_transfer_region_type(obj_prop_xx, PDC_REGION_LOCAL);
  ////////////////////////////////////////////////////

  PDCprop_set_obj_dims(obj_prop_xx, 1, dims);
  PDCprop_set_obj_type(obj_prop_xx, PDC_FLOAT);
  PDCprop_set_obj_time_step(obj_prop_xx, 0);
  PDCprop_set_obj_user_id(obj_prop_xx, getuid());
  PDCprop_set_obj_app_name(obj_prop_xx, (char *)std::string("VPICIO").c_str());
  PDCprop_set_obj_tags(obj_prop_xx, (char *)std::string("tag0=1").c_str());

  pdcid_t obj_prop_yy = PDCprop_obj_dup(obj_prop_xx);
  PDCprop_set_obj_type(obj_prop_yy, PDC_FLOAT);

  pdcid_t obj_prop_zz = PDCprop_obj_dup(obj_prop_xx);
  PDCprop_set_obj_type(obj_prop_zz, PDC_FLOAT);

  pdcid_t obj_prop_pxx = PDCprop_obj_dup(obj_prop_xx);
  PDCprop_set_obj_type(obj_prop_pxx, PDC_FLOAT);

  pdcid_t obj_prop_pyy = PDCprop_obj_dup(obj_prop_xx);
  PDCprop_set_obj_type(obj_prop_pyy, PDC_FLOAT);

  pdcid_t obj_prop_pzz = PDCprop_obj_dup(obj_prop_xx);
  PDCprop_set_obj_type(obj_prop_pzz, PDC_FLOAT);

  pdcid_t obj_prop_id11 = PDCprop_obj_dup(obj_prop_xx);
  PDCprop_set_obj_type(obj_prop_id11, PDC_INT);

  pdcid_t obj_prop_id22 = PDCprop_obj_dup(obj_prop_xx);
  PDCprop_set_obj_type(obj_prop_id22, PDC_INT);

  for(int timestep = 0; timestep < 1; timestep++)
  {
    PDCprop_set_obj_time_step(obj_prop_xx, timestep);
    PDCprop_set_obj_time_step(obj_prop_yy, timestep);
    PDCprop_set_obj_time_step(obj_prop_zz, timestep);
    PDCprop_set_obj_time_step(obj_prop_pxx, timestep);
    PDCprop_set_obj_time_step(obj_prop_pyy, timestep);
    PDCprop_set_obj_time_step(obj_prop_pzz, timestep);
    PDCprop_set_obj_time_step(obj_prop_id11, timestep);
    PDCprop_set_obj_time_step(obj_prop_id22, timestep);


    pdcid_t obj_xx = PDCobj_create_mpi(container_id,   ("obj-var-xx"+std::to_string(timestep)).c_str(), obj_prop_xx, 0, comm);
    pdcid_t obj_yy = PDCobj_create_mpi(container_id,   ("obj-var-yy"+std::to_string(timestep)).c_str(), obj_prop_yy, 0, comm);
    pdcid_t obj_zz = PDCobj_create_mpi(container_id,   ("obj-var-zz"+std::to_string(timestep)).c_str(), obj_prop_zz, 0, comm);
    pdcid_t obj_pxx = PDCobj_create_mpi(container_id,  ("obj-var-pxx"+std::to_string(timestep)).c_str(), obj_prop_pxx, 0, comm);
    pdcid_t obj_pyy = PDCobj_create_mpi(container_id,  ("obj-var-pyy"+std::to_string(timestep)).c_str(), obj_prop_pyy, 0, comm);
    pdcid_t obj_pzz = PDCobj_create_mpi(container_id,  ("obj-var-pzz"+std::to_string(timestep)).c_str(), obj_prop_pzz, 0, comm);
    pdcid_t obj_id11 = PDCobj_create_mpi(container_id, ("id11"+std::to_string(timestep)).c_str(), obj_prop_id11, 0, comm);
    pdcid_t obj_id22 = PDCobj_create_mpi(container_id, ("id22"+std::to_string(timestep)).c_str(), obj_prop_id22, 0, comm);

    int ndim = 1;
    uint64_t *offset = (uint64_t *)malloc(sizeof(uint64_t) * ndim);
    offset[0] = 0;

    uint64_t *offset_remote = (uint64_t *)malloc(sizeof(uint64_t) * ndim);
    offset_remote[0] = rank * numparticles;

    uint64_t *mysize = (uint64_t *)malloc(sizeof(uint64_t) * ndim);
    mysize[0] = numparticles;

    //////////////////////////////////////////////////////////////////////////////////////////
    // register
    // PDC_API_CALL( PDCobj_transform_register("pdc_transform_increment", obj_id11, 0, INCR_STATE, PDC_DATA_MAP, DATA_OUT) );
    //////////////////////////////////////////////////////////////////////////////////////////

    // create a region
    pdcid_t region_x   = PDCregion_create(ndim, offset, mysize);
    pdcid_t region_y   = PDCregion_create(ndim, offset, mysize);
    pdcid_t region_z   = PDCregion_create(ndim, offset, mysize);
    pdcid_t region_px  = PDCregion_create(ndim, offset, mysize);
    pdcid_t region_py  = PDCregion_create(ndim, offset, mysize);
    pdcid_t region_pz  = PDCregion_create(ndim, offset, mysize);
    pdcid_t region_id1 = PDCregion_create(ndim, offset, mysize);
    pdcid_t region_id2 = PDCregion_create(ndim, offset, mysize);

    pdcid_t region_xx   = PDCregion_create(ndim, offset_remote, mysize);
    pdcid_t region_yy   = PDCregion_create(ndim, offset_remote, mysize);
    pdcid_t region_zz   = PDCregion_create(ndim, offset_remote, mysize);
    pdcid_t region_pxx  = PDCregion_create(ndim, offset_remote, mysize);
    pdcid_t region_pyy  = PDCregion_create(ndim, offset_remote, mysize);
    pdcid_t region_pzz  = PDCregion_create(ndim, offset_remote, mysize);
    pdcid_t region_id11 = PDCregion_create(ndim, offset_remote, mysize);
    pdcid_t region_id22 = PDCregion_create(ndim, offset_remote, mysize);

    PDC_API_CALL( PDCbuf_map_transform_register("pdc_cusz_compress:libpdc_transform_cusz.so", &d_x[0], region_x, obj_xx, region_xx, 0, INCR_STATE, DATA_OUT) );

    PDC_API_CALL( PDCbuf_obj_map(&d_x[0], PDC_FLOAT, region_x, obj_xx, region_xx) );
    PDC_API_CALL( PDCbuf_obj_map(&d_y[0], PDC_FLOAT, region_y, obj_yy, region_yy) );
    PDC_API_CALL( PDCbuf_obj_map(&d_z[0], PDC_FLOAT, region_z, obj_zz, region_zz) );
    PDC_API_CALL( PDCbuf_obj_map(&d_px[0], PDC_FLOAT, region_px, obj_pxx, region_pxx) );
    PDC_API_CALL( PDCbuf_obj_map(&d_py[0], PDC_FLOAT, region_py, obj_pyy, region_pyy) );
    PDC_API_CALL( PDCbuf_obj_map(&d_pz[0], PDC_FLOAT, region_pz, obj_pzz, region_pzz) );
    PDC_API_CALL( PDCbuf_obj_map(&d_id1[0], PDC_INT, region_id1, obj_id11, region_id11) );
    PDC_API_CALL( PDCbuf_obj_map(&d_id2[0], PDC_INT, region_id2, obj_id22, region_id22) );

    PDC_API_CALL( PDCreg_obtain_lock(obj_xx, region_xx, PDC_WRITE, PDC_NOBLOCK) );
    PDC_API_CALL( PDCreg_obtain_lock(obj_yy, region_yy, PDC_WRITE, PDC_NOBLOCK) );
    PDC_API_CALL( PDCreg_obtain_lock(obj_zz, region_zz, PDC_WRITE, PDC_NOBLOCK) );
    PDC_API_CALL( PDCreg_obtain_lock(obj_pxx, region_pxx, PDC_WRITE, PDC_NOBLOCK) );
    PDC_API_CALL( PDCreg_obtain_lock(obj_pyy, region_pyy, PDC_WRITE, PDC_NOBLOCK) );
    PDC_API_CALL( PDCreg_obtain_lock(obj_pzz, region_pzz, PDC_WRITE, PDC_NOBLOCK) );
    PDC_API_CALL( PDCreg_obtain_lock(obj_id11, region_id11, PDC_WRITE, PDC_NOBLOCK) );
    PDC_API_CALL( PDCreg_obtain_lock(obj_id22, region_id22, PDC_WRITE, PDC_NOBLOCK) );

    // compute
    for (uint64_t i = 0; i < numparticles; i++) {
      d_id1[i] = timestep * i;
      d_id2[i] = timestep * i * 2;
      d_x[i]   = timestep * uniform_random_number() * x_dim;
      d_y[i]   = timestep * uniform_random_number() * y_dim;
      d_z[i]   = timestep * ((float)id1[i] / numparticles) * z_dim;
      d_px[i]  = timestep * uniform_random_number() * x_dim;
      d_py[i]  = timestep * uniform_random_number() * y_dim;
      d_pz[i]  = timestep * ((float)id2[i] / numparticles) * z_dim;
    }

    // CUDA_RUNTIME_API_CALL( hipMemcpy(d_x,   x, numparticles*sizeof(float), hipMemcpyHostToDevice) );
    // CUDA_RUNTIME_API_CALL( hipMemcpy(d_y,   y, numparticles*sizeof(float), hipMemcpyHostToDevice) );
    // CUDA_RUNTIME_API_CALL( hipMemcpy(d_z,   z, numparticles*sizeof(float), hipMemcpyHostToDevice) );
    // CUDA_RUNTIME_API_CALL( hipMemcpy(d_px,  px, numparticles*sizeof(float), hipMemcpyHostToDevice) );
    // CUDA_RUNTIME_API_CALL( hipMemcpy(d_py,  py, numparticles*sizeof(float), hipMemcpyHostToDevice) );
    // CUDA_RUNTIME_API_CALL( hipMemcpy(d_pz,  pz, numparticles*sizeof(float), hipMemcpyHostToDevice) );
    // CUDA_RUNTIME_API_CALL( hipMemcpy(d_id1, id1, numparticles*sizeof(int), hipMemcpyHostToDevice) );
    // CUDA_RUNTIME_API_CALL( hipMemcpy(d_id2, id2, numparticles*sizeof(int), hipMemcpyHostToDevice) );

    PDC_API_CALL( PDCreg_release_lock(obj_xx, region_xx, PDC_WRITE) );
    PDC_API_CALL( PDCreg_release_lock(obj_yy, region_yy, PDC_WRITE) );
    PDC_API_CALL( PDCreg_release_lock(obj_zz, region_zz, PDC_WRITE) );
    PDC_API_CALL( PDCreg_release_lock(obj_pxx, region_pxx, PDC_WRITE) );
    PDC_API_CALL( PDCreg_release_lock(obj_pyy, region_pyy, PDC_WRITE) );
    PDC_API_CALL( PDCreg_release_lock(obj_pzz, region_pzz, PDC_WRITE) );
    PDC_API_CALL( PDCreg_release_lock(obj_id11, region_id11, PDC_WRITE) );
    PDC_API_CALL( PDCreg_release_lock(obj_id22, region_id22, PDC_WRITE) );

    PDC_API_CALL( PDCbuf_obj_unmap(obj_xx, region_xx) );
    PDC_API_CALL( PDCbuf_obj_unmap(obj_yy, region_yy) );
    PDC_API_CALL( PDCbuf_obj_unmap(obj_zz, region_zz) );
    PDC_API_CALL( PDCbuf_obj_unmap(obj_pxx, region_pxx) );
    PDC_API_CALL( PDCbuf_obj_unmap(obj_pyy, region_pyy) );
    PDC_API_CALL( PDCbuf_obj_unmap(obj_pzz, region_pzz) );
    PDC_API_CALL( PDCbuf_obj_unmap(obj_id11, region_id11) );
    PDC_API_CALL( PDCbuf_obj_unmap(obj_id22, region_id22) );

    PDC_API_CALL(PDCregion_close(region_x));
    PDC_API_CALL(PDCregion_close(region_y));
    PDC_API_CALL(PDCregion_close(region_z));
    PDC_API_CALL(PDCregion_close(region_px));
    PDC_API_CALL(PDCregion_close(region_py));
    PDC_API_CALL(PDCregion_close(region_pz));
    PDC_API_CALL(PDCregion_close(region_id1));
    PDC_API_CALL(PDCregion_close(region_id2));

    PDC_API_CALL(PDCregion_close(region_xx));
    PDC_API_CALL(PDCregion_close(region_yy));
    PDC_API_CALL(PDCregion_close(region_zz));
    PDC_API_CALL(PDCregion_close(region_pxx));
    PDC_API_CALL(PDCregion_close(region_pyy));
    PDC_API_CALL(PDCregion_close(region_pzz));
    PDC_API_CALL(PDCregion_close(region_id11));
    PDC_API_CALL(PDCregion_close(region_id22));

    PDC_API_CALL(PDCobj_close(obj_xx));
    PDC_API_CALL(PDCobj_close(obj_yy));
    PDC_API_CALL(PDCobj_close(obj_zz));
    PDC_API_CALL(PDCobj_close(obj_pxx));
    PDC_API_CALL(PDCobj_close(obj_pyy));
    PDC_API_CALL(PDCobj_close(obj_pzz));
    PDC_API_CALL(PDCobj_close(obj_id11));
    PDC_API_CALL(PDCobj_close(obj_id22));

    free(offset);
    free(offset_remote);
    free(mysize);

#ifdef ENABLE_MPI
    MPI_Barrier(MPI_COMM_WORLD);
#endif
  }

  PDC_API_CALL(PDCprop_close(obj_prop_xx));
  PDC_API_CALL(PDCprop_close(obj_prop_yy));
  PDC_API_CALL(PDCprop_close(obj_prop_zz));
  PDC_API_CALL(PDCprop_close(obj_prop_pxx));
  PDC_API_CALL(PDCprop_close(obj_prop_pyy));
  PDC_API_CALL(PDCprop_close(obj_prop_pzz));
  PDC_API_CALL(PDCprop_close(obj_prop_id11));
  PDC_API_CALL(PDCprop_close(obj_prop_id22));

  ///////////////////////////////////
  // printf("call wait_my_rpc()\n");
  // wait_my_rpc();
  // printf("finish wait_my_rpc()\n");
  ///////////////////////////////////

  PDC_API_CALL(PDCcont_close(container_id));
  PDC_API_CALL(PDCprop_close(container_prop));
  PDC_API_CALL(PDCclose(pdc_id));

  free(x);
  free(y);
  free(z);
  free(px);
  free(py);
  free(pz);
  free(id1);
  free(id2);

  nvtxRangePop();

#ifdef ENABLE_MPI
  MPI_Finalize();
#endif

  return 0;
}
